#include "hip/hip_runtime.h"
#include "argsort.cuh"

template<typename T>
static inline __device__ void ggml_cuda_swap(T & a, T & b) {
    T tmp = a;
    a = b;
    b = tmp;
}

template<ggml_sort_order order>
static __global__ void k_argsort_f32_i32(const float * x, int * dst, const int ncols) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.y;

    if (col >= ncols) return;

    const float * x_row = x + row * ncols;
    int * dst_row = dst + row * ncols;

    // initialize indices
    if (col < ncols) {
        dst_row[col] = col;
    }
    __syncthreads();

    for (int k = 2; k <= ncols; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (order == GGML_SORT_ORDER_ASC ? x_row[dst_row[col]] > x_row[dst_row[ixj]] : x_row[dst_row[col]] < x_row[dst_row[ixj]]) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                } else {
                    if (order == GGML_SORT_ORDER_ASC ? x_row[dst_row[col]] < x_row[dst_row[ixj]] : x_row[dst_row[col]] > x_row[dst_row[ixj]]) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }
}

static void argsort_f32_i32_cuda(const float * x, int * dst, const int ncols, const int nrows, ggml_sort_order order, hipStream_t stream) {
    // bitonic sort requires ncols to be power of 2
    GGML_ASSERT((ncols & (ncols - 1)) == 0);

    const dim3 block_dims(ncols, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    if (order == GGML_SORT_ORDER_ASC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_ASC><<<block_nums, block_dims, 0, stream>>>(x, dst, ncols);
    } else if (order == GGML_SORT_ORDER_DESC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_DESC><<<block_nums, block_dims, 0, stream>>>(x, dst, ncols);
    } else {
        GGML_ASSERT(false);
    }
}

void ggml_cuda_op_argsort(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    enum ggml_sort_order order = (enum ggml_sort_order) dst->op_params[0];

    argsort_f32_i32_cuda(src0_d, (int *)dst_d, ncols, nrows, order, stream);
}
