#include "hip/hip_runtime.h"
#include "quantize.cuh"

static __global__ void quantize_q8_1(const float * __restrict__ x, void * __restrict__ vy, const int kx, const int kx_padded) {
    const int ix = blockDim.x*blockIdx.x + threadIdx.x;

    if (ix >= kx_padded) {
        return;
    }

    const int iy = blockDim.y*blockIdx.y + threadIdx.y;

    const int i_padded = iy*kx_padded + ix;

    block_q8_1 * y = (block_q8_1 *) vy;

    const int ib = i_padded / QK8_1; // block index
    const int iqs = i_padded % QK8_1; // quant index

    const float xi = ix < kx ? x[iy*kx + ix] : 0.0f;
    float amax = fabsf(xi);
    float sum = xi;

    amax = warp_reduce_max(amax);
    sum = warp_reduce_sum(sum);

    const float d = amax / 127;
    const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    y[ib].qs[iqs] = q;

    if (iqs > 0) {
        return;
    }

    reinterpret_cast<half&>(y[ib].ds.x) = d;
    reinterpret_cast<half&>(y[ib].ds.y) = sum;
}

void quantize_row_q8_1_cuda(const float * x, void * vy, const int kx, const int ky, const int kx_padded, hipStream_t stream) {
    const int block_num_x = (kx_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
    const dim3 num_blocks(block_num_x, ky, 1);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(x, vy, kx, kx_padded);
}

